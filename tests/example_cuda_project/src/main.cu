#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

__global__ void sleep_kernel(int64_t num_cycles, int64_t clock_rate) {
    int64_t start = clock64();
    int64_t next_tick = start + clock_rate;
    while (clock64() - start < num_cycles) {
        if (clock64() >= next_tick) {
            next_tick += clock_rate;
            printf("Slept for %f seconds\n", (clock64() - start) / static_cast<float>(clock_rate));
        }
        // sleep
    }
}

int main() {
    int device_count;
    hipGetDeviceCount(&device_count);
    std::cout << "Number of CUDA devices: " << device_count << std::endl;

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    int64_t clock_rate = prop.clockRate * 1000;
    float sleep_seconds = 10.0f;
    int64_t num_cycles = static_cast<int64_t>(sleep_seconds * clock_rate);

    sleep_kernel<<<1, 1>>>(num_cycles, clock_rate);
    hipDeviceSynchronize();
    
    std::cout << "Slept for " << sleep_seconds << " seconds" << std::endl;
    return 0;
}